
#include <hip/hip_runtime.h>
#include <stdio.h>





extern "C" {


  void cudamallochost(void **hostptr, int* size);

}






//Fortran entry for allocating pinned memory
void cudamallochost(void **hostptr, int *p_size) {
  hipError_t err;

  err = hipHostAlloc((void **)hostptr, (*p_size)*sizeof(double), hipHostMallocMapped);
  if (err != hipSuccess) fprintf(stderr, "Error in cudaHostAlloc: %s\n", hipGetErrorString( err ));
}
