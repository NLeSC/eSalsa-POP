
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * This file contains the necessary functions for interfacing Fortran with the CUDA runtime API
 *
 *
 */



extern "C" {


  void cudamallochost(void **hostptr, int *p_size);

}



int cuda_initialized = 0;

void cuda_init() {
  hipSetDeviceFlags(hipDeviceMapHost);
  hipSetDevice(0);
  hipDeviceSetCacheConfig(hipFuncCachePreferL1);
  hipDeviceSynchronize();


}


//Fortran entry for allocating pinned memory
void cudamallochost(void **hostptr, int *p_size) {
  if (!cuda_initialized) {
    cuda_initialized = 1;
    cuda_init();
  }
 
  hipError_t err;

  err = hipHostAlloc((void **)hostptr, (*p_size)*sizeof(double), hipHostMallocMapped);
  if (err != hipSuccess) fprintf(stderr, "Error in cudaHostAlloc: %s\n", hipGetErrorString( err ));
}
