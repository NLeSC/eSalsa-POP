
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * This file contains the necessary functions for interfacing Fortran with the CUDA runtime API
 *
 *
 */



extern "C" {

  void cuda_init(int *pmy_task);

  void cudamallochost(void **hostptr, int *p_size);

}



int my_task;
int cuda_initialized = 0;

//Fortran entry for initializing CUDA
void cuda_init(int *pmy_task) {
  if (cuda_initialized == 0) {
    cuda_initialized = 1;
    my_task = *pmy_task;
    int deviceCount = 0;

    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) fprintf(stderr, "Error in cuda initialization: %s\n", hipGetErrorString( err ));

    if (deviceCount < 1) {
      fprintf(stderr,"Error: less than 1 cuda capable device detected proc=%d\n", my_task);
    }

    int dev = my_task % deviceCount;
    //fprintf(stdout,"Process %d: using CUDA device %d\n",my_task,dev);

    hipSetDeviceFlags(hipDeviceMapHost);
    hipSetDevice(dev);

    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    hipDeviceSynchronize();
  }
}


//Fortran entry for allocating pinned memory
void cudamallochost(void **hostptr, int *p_size) {
  if (!cuda_initialized) {
    printf("Error: cudamallochost called before cuda_init\n");
  }
 
  hipError_t err;

  err = hipHostAlloc((void **)hostptr, (*p_size)*sizeof(double), hipHostMallocMapped);
  if (err != hipSuccess) fprintf(stderr, "Error in cudaHostAlloc: %s\n", hipGetErrorString( err ));
}
